#include "hip/hip_runtime.h"
#pragma once
#include "rayHelpers.cu"



hipError_t cuda();
__global__ void kernel(){
  
}



__device__ bool intersectsSphere(const float3 &origin, const float3& dir,  const sphereInfo& info, float &t) {

		float t0, t1; // solutions for t if the ray intersects 

		float3 L = info.pos - origin;
		float tca = dot(dir, L);
		 //if (tca < 0) return false;
		float d2 = dot(L,L) - tca * tca;
		if (d2 > info.rad2) return false;
		float thc = sqrt(info.rad2 - d2);
		t0 = tca - thc;
		t1 = tca + thc;

		if (t0 > t1) {
			float temp = t0;
			t0 = t1;
			t1 = temp;
		}

		if (t0 < 0) {
			t0 = t1; // if t0 is negative, let's use t1 instead 
			if (t0 < 0) return false; // both t0 and t1 are negative 
		}
		t = t0;
		return true;
}

// plane normal, plane point, ray start, ray dir, point along line
__device__ bool intersectPlane(const planeInfo& p, const float3& l0, const float3& l, float& t)
{
	// assuming vectors are all normalized
	float denom = dot(p.normal, l);
	if (denom > 1e-6) {
		float3 p0l0 = p.point - l0;
		t = dot(p0l0, p.normal) / denom;
		return (t >= 0);
	}
	return false;
}


__device__ void fresnel(const float3& I, const float3& N, const float& ior, float& kr)
{
	float cosi = clamp(-1, 1, dot(I, N));
	float etai = 1, etat = ior;
	if (cosi > 0) { float temp = etai; etai = etat; etat = temp;}
	// Compute sini using Snell's law
	float sint = etai / etat * sqrtf(max(0.f, 1 - cosi * cosi));
	// Total internal reflection
	if (sint >= 1) {
		kr = 1;
	}
	else {
		float cost = sqrtf(max(0.f, 1 - sint * sint));
		cosi = abs(cosi);
		float Rs = ((etat * cosi) - (etai * cost)) / ((etat * cosi) + (etai * cost));
		float Rp = ((etai * cosi) - (etat * cost)) / ((etai * cosi) + (etat * cost));
		kr = (Rs * Rs + Rp * Rp) / 2;
	}
}


__device__ float3 refract(const float3& I, const float3& N, const float& ior)
{
	float cosi = clamp(-1, 1, dot(I, N));
	float etai = 1, etat = ior;
	float3 n = N;
	if (cosi < 0) { cosi = -cosi; }
	else { float temp = etai; etai = etat; etat = temp; n = inverse(N); }
	float eta = etai / etat;
	float k = 1 - eta * eta * (1 - cosi * cosi);
	return eta * I + (eta * cosi - sqrtf(k)) * n;
}

__device__ float3 reflect(const float3& I, const float3& N)
{
	return I - 2 * dot(I, N) * N;
}

struct hitInfo {
	int objectIndex = -1;
	float3 pos;
	float3 normal;

};

#define LIGHT_POS make_float3(0,5,20)


__device__ hitInfo getHit(float3 currRayPos, float3 currRayDir, const float& currTime, const objectInfo* objects, int numObjects) {
	float closestDist = 1000000;
	float3 normal;
	hitInfo toReturn;
	int closestObjectIndex = -1;


	for (int i = 0; i < numObjects; i++) {
		const objectInfo& curr = objects[i];
		float currDist;

		switch (curr.s) {
		case plane: {
			planeInfo* p1 = (planeInfo*)curr.shapeData;
			if (intersectPlane(*p1, currRayPos, currRayDir, currDist) && currDist < closestDist) {
				closestDist = currDist;
				closestObjectIndex = i;

				normal = p1->normal;
			}

			break;
		}
		case sphere: {
			sphereInfo* s1 = (sphereInfo*)curr.shapeData;
			if (intersectsSphere(currRayPos, currRayDir, *s1, currDist) && currDist < closestDist) {
				closestDist = currDist;
				closestObjectIndex = i;

				float3 nextPos = currRayPos + currDist * currRayDir;
				normal = normalize(nextPos - s1->pos);

			}
			break;
		}
		}
	}

	toReturn.objectIndex = closestObjectIndex;
	toReturn.normal = normal;
	toReturn.pos = currRayPos + closestDist * currRayDir;
	return toReturn;
}


__device__ float getShadowTerm(const float3 originalPos, const float currTime, const objectInfo* objects, int numObjects) {
	float3 toLightVec = normalize(LIGHT_POS - originalPos);
	hitInfo hit = getHit(originalPos, toLightVec, currTime, objects, numObjects);

	if (hit.objectIndex == -1 || length(hit.pos - originalPos) > length(originalPos - LIGHT_POS)) {
		return 1.;
	}
	return objects[hit.objectIndex].refractivity * 0.8 + 0.2;
	//while (length(currPos - LIGHT_POS) > 0.1) {
	//
	//}

}

__device__ float3 trace(const float3 currRayPos, const float3 currRayDir, int remainingDepth, const float currTime, objectInfo *objects, int numObjects) {
	if (remainingDepth <= 0) {
		return make_float3(0,0,0);
	}

	hitInfo hit = getHit(currRayPos, currRayDir, currTime, objects, numObjects);

	if (hit.objectIndex == -1) {
		return make_float3(0,0,0);
	}
	else {
		objectInfo currObject = objects[hit.objectIndex];
		float3 reflected = make_float3(0, 0, 0);
		float3 refracted = make_float3(0, 0, 0);
		float3 nextPos = hit.pos;
		float3 normal = hit.normal;

		float extraReflection = 0;
		float3 bias = 0.001 * normal;
		if (currObject.refractivity > 0.) {
			float kr;
			bool outside = dot(currRayDir, normal) < 0;
			fresnel(currRayDir, normal, outside? currObject.refractiveIndex : 1 / currObject.refractiveIndex, kr);


			if (kr < 1) {
				float3 refractionDirection = normalize(refract(currRayDir, normal, currObject.refractiveIndex));
				float3 refractionRayOrig = outside ? nextPos - bias : nextPos + bias;
				refracted = currObject.refractivity *(1-kr)* trace(refractionRayOrig, refractionDirection, remainingDepth - 1, currTime, objects, numObjects);
			}
			extraReflection = min(1.,kr) * currObject.refractivity;

		}
		if (currObject.reflectivity + extraReflection > 0.) {
			float3 reflectDir = reflect(currRayDir, normal);
			reflected = (currObject.reflectivity + extraReflection )* trace(nextPos + bias, reflectDir, remainingDepth - 1, currTime, objects, numObjects);
		}
		float3 color = (1 - currObject.reflectivity - extraReflection - currObject.refractivity) * currObject.color;
		return 1000 * (1 / powf(length(nextPos - LIGHT_POS), 2)) * getShadowTerm(nextPos + bias, currTime, objects, numObjects) * color + reflected + refracted;
	}

}

struct inputStruct {
	float currPosX;
	float currPosY;
	float currPosZ;

	float forwardX;
	float forwardY;
	float forwardZ;

	float upX;
	float upY;
	float upZ;

};

__global__ void
cudaRender(unsigned int *g_odata, int imgw, int imgh, float currTime, inputStruct input)
{
	extern __shared__ uchar4 sdata[];

	int tx = threadIdx.x;
	int ty = threadIdx.y;
	int bw = blockDim.x;
	int bh = blockDim.y;
	int x = blockIdx.x*bw + tx;
	int y = blockIdx.y*bh + ty;

	float3 forwardV = make_float3(input.forwardX, input.forwardY, input.forwardZ);
	float3 upV = make_float3(input.upX, input.upY, input.upZ);
	float3 rightV = normalize(cross(upV,forwardV));

	float sizeFarPlane = 10;
	float sizeNearPlane = sizeFarPlane*0.5;
	float3 origin = make_float3(input.currPosX, input.currPosY, input.currPosZ);
	float distFarPlane = 4;
	float distFirstPlane = distFarPlane *0.5;

	float3 center = make_float3(imgw / 2.0, imgh / 2.0, 0.);
	float3 distFromCenter = ((x - center.x) / imgw) * rightV + ((center.y - y) / imgh) * upV;
	float3 firstPlanePos = (sizeNearPlane*distFromCenter) + origin + (distFirstPlane * forwardV);
	float3 secondPlanePos = (sizeFarPlane * distFromCenter) + (distFarPlane * forwardV) + origin;




	float3 dirVector = normalize(secondPlanePos - firstPlanePos);

	sphereInfo s1 = make_sphereInfo(make_float3(sin(currTime) * 2.0, -3, cos(currTime) * 2 - 15), 1);
	sphereInfo s2 = make_sphereInfo(make_float3(-15, -4, -15), 4);
	sphereInfo s3 = make_sphereInfo(make_float3(2, 4, -40), 8);
	sphereInfo s4 = make_sphereInfo(make_float3(sin(currTime * 0.2)*6 + 4, 1,  cos(currTime*0.2) * 5 - 10), 3);
	planeInfo p1 = make_planeInfo(make_float3(0, -4.0, 0), make_float3(0, -1, 0));
	planeInfo p2 = make_planeInfo(make_float3(0, 50.0, 0), make_float3(0, 1, 0));
	planeInfo p3 = make_planeInfo(make_float3(0, 0.0, -70), make_float3(0, 0, -1));
	planeInfo p4 = make_planeInfo(make_float3(70, 0, 0), make_float3(1, 0, 0));

	objectInfo objects[10];
	objects[0] = make_objectInfo(sphere, &s1, 0.0, make_float3(1, 0, 0),0,0);
	objects[1] = make_objectInfo(sphere, &s2, 0.5, make_float3(0, 1, 0),0.0,1.5);
	objects[2] = make_objectInfo(plane, &p1, 0.2, make_float3(0, 1, 1),0,0);
	objects[3] = make_objectInfo(sphere, &s3, 0.7, make_float3(1, 1, 1), 0,0);
	objects[4] = make_objectInfo(plane, &p2, 0.0, make_float3(1, 1, 1), 0,0);
	objects[5] = make_objectInfo(sphere, &s4, 0.0, make_float3(1, 1, 1), 0.9,1.5);
	objects[6] = make_objectInfo(plane, &p3, 0.5, make_float3(0, 1, 0), 0,0);
	objects[7] = make_objectInfo(plane, &p4, 0.5, make_float3(0, 1, 0), 0,0);

	float3 out = 255*trace(firstPlanePos, dirVector, 1, currTime, objects, 8);


	g_odata[y * imgw + x] = rgbToInt(out.x, out.y, out.z);
}
extern "C" void
launch_cudaRender(dim3 grid, dim3 block, int sbytes, unsigned int *g_odata, int imgw, int imgh, float currTime,inputStruct input)
{

	cudaRender << < grid, block, sbytes >> >(g_odata, imgw, imgh, currTime, input);
}

