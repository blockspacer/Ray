#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "vector_functions.h"
#include "common_functions.h"
#include "math_functions.h"
#include <stdlib.h>
#include <stdio.h>
#include "rayHelpers.cu"

hipError_t cuda();

__global__ void kernel(){
  
}

// clamp x to range [a, b]
__device__ float clamp(float x, float a, float b)
{
	return max(a, min(b, x));
}

// convert floating point rgb color to 8-bit integer
__device__ int rgbToInt(float r, float g, float b)
{
	r = clamp(r, 0.0f, 255.0f);
	g = clamp(g, 0.0f, 255.0f);
	b = clamp(b, 0.0f, 255.0f);
	return (int(b) << 16) | (int(g) << 8) | int(r);
}

__device__ float3 operator+(const float3& a, const float3& b) {
	return make_float3(a.x + b.x, a.y + b.y, a.z + b.z);
}

__device__ float3 operator*(const float3& a, const float3& b) {
	return make_float3(a.x * b.x, a.y * b.y, a.z * b.z);
}


__device__ float3 operator*(const float& a, const float3& b) {
	return make_float3(a * b.x, a * b.y, a * b.z);
}

__device__ float3 operator-(const float3& a, const float3& b) {
	return make_float3(a.x - b.x, a.y - b.y, a.z - b.z);
}

inline __device__  float dot(float3 v1, float3 v2)
{
	return v1.x * v2.x + v1.y * v2.y + v1.z * v2.z;
}
inline __device__ float length(float3 v)
{
	return sqrtf(v.x * v.x + v.y * v.y + v.z * v.z);
}


inline __device__ float3 inverse(float3 v)
{
	return make_float3(-v.x, -v.y, -v.z);
}


inline __device__ float3 normalize(float3 v)
{
	float invLen = /*1.0f / length(v);*/ 1/sqrtf(dot(v, v));
	return invLen * v;
}


struct sphereInfo {
	float3 pos;
	float rad;
	float rad2;
};

__device__ sphereInfo make_sphereInfo(float3 pos, float rad) {
	sphereInfo s;
	s.pos = pos;
	s.rad = rad;
	s.rad2 = rad * rad;
	return s;
}


struct planeInfo {
	float3 point;
	float3 normal;
};

__device__ planeInfo make_planeInfo(float3 point, float3 normal) {
	planeInfo p;
	p.point = point;
	p.normal = normal;
	return p;
}

enum shape{sphere, plane};

struct objectInfo {
	shape s;
	void* shapeData;
	float reflectivity;
	float refractivity;
	float refractiveIndex;
	float3 color;
};

__device__ objectInfo make_objectInfo(shape s, void* shapeData, float reflectivity, float3 color, float refractivity, float refractiveIndex) {
	objectInfo o;
	o.s = s;
	o.shapeData = shapeData;
	o.reflectivity = reflectivity;
	o.color = color;
	o.refractivity = refractivity;
	o.refractiveIndex = refractiveIndex;
	return o;
}


__device__ bool intersectsSphere(const float3 &origin, const float3& dir,  const sphereInfo& info, float &t) {

		float t0, t1; // solutions for t if the ray intersects 

		float3 L = info.pos - origin;
		float tca = dot(dir, L);
		// if (tca < 0) return false;
		float d2 = dot(L,L) - tca * tca;
		if (d2 > info.rad2) return false;
		float thc = sqrt(info.rad2 - d2);
		t0 = tca - thc;
		t1 = tca + thc;

		if (t0 > t1) {
			float temp = t0;
			t0 = t1;
			t1 = temp;
		}

		if (t0 < 0) {
			t0 = t1; // if t0 is negative, let's use t1 instead 
			if (t0 < 0) return false; // both t0 and t1 are negative 
		}
		t = t0;
		return true;
}

// plane normal, plane point, ray start, ray dir, point along line
__device__ bool intersectPlane(const planeInfo& p, const float3& l0, const float3& l, float& t)
{
	// assuming vectors are all normalized
	float denom = dot(p.normal, l);
	if (denom > 1e-6) {
		float3 p0l0 = p.point - l0;
		t = dot(p0l0, p.normal) / denom;
		return (t >= 0);
	}
	return false;
}


__device__ void fresnel(const float3& I, const float3& N, const float& ior, float& kr)
{
	float cosi = clamp(-1, 1, dot(I, N));
	float etai = 1, etat = ior;
	if (cosi > 0) { float temp = etai; etai = etat; etat = temp;}
	// Compute sini using Snell's law
	float sint = etai / etat * sqrtf(max(0.f, 1 - cosi * cosi));
	// Total internal reflection
	if (sint >= 1) {
		kr = 1;
	}
	else {
		float cost = sqrtf(max(0.f, 1 - sint * sint));
		cosi = abs(cosi);
		float Rs = ((etat * cosi) - (etai * cost)) / ((etat * cosi) + (etai * cost));
		float Rp = ((etai * cosi) - (etat * cost)) / ((etai * cosi) + (etat * cost));
		kr = (Rs * Rs + Rp * Rp) / 2;
	}
	// As a consequence of the conservation of energy, transmittance is given by:
	// kt = 1 - kr;
}


__device__ float3 refract(const float3& I, const float3& N, const float& ior)
{
	float cosi = clamp(-1, 1, dot(I, N));
	float etai = 1, etat = ior;
	float3 n = N;
	if (cosi < 0) { cosi = -cosi; }
	else { float temp = etai; etai = etat; etat = temp; n = inverse(N); }
	float eta = etai / etat;
	float k = 1 - eta * eta * (1 - cosi * cosi);
	return eta * I + (eta * cosi - sqrtf(k)) * n;
}

__device__ float3 reflect(const float3& I, const float3& N)
{
	return I - 2 * dot(I, N) * N;
}

__device__ float3 trace(float3 currRayPos, float3 currRayDir, int remainingDepth, const float &currTime, objectInfo objects[], int numObjects) {
	if (remainingDepth <= 0) {
		return make_float3(0,0,0);
	}

	float closestDist = 1000000;
	float3 normal;
	int closestObjectIndex = -1;

	//for (int j = 0; j < 100; j++) {
		for (int i = 0; i < numObjects; i++) {
			const objectInfo& curr = objects[i];
			float currDist;

			switch (curr.s) {
			case plane: {
				planeInfo* p1 = (planeInfo*)curr.shapeData;
				if (intersectPlane(*p1, currRayPos, currRayDir, currDist) && currDist < closestDist) {
					closestDist = currDist;
					closestObjectIndex = i;

					normal = p1->normal;
				}

				break;
			}
			case sphere: {
				sphereInfo* s1 = (sphereInfo*)curr.shapeData;
				if (intersectsSphere(currRayPos, currRayDir, *s1, currDist) && currDist < closestDist) {
					closestDist = currDist;
					closestObjectIndex = i;

					float3 nextPos = currRayPos + currDist * currRayDir;
					normal = normalize(nextPos - s1->pos);

				}
				break;
			}
			}
		}
	//}

	if (closestObjectIndex == -1) {
		return make_float3(0,0,0);
	}
	else {
		objectInfo currObject = objects[closestObjectIndex];
		float3 reflected = make_float3(0, 0, 0);
		float3 refracted = make_float3(0, 0, 0);
		float3 nextPos = currRayPos + closestDist * currRayDir;

		float extraReflection = 0;
		if (currObject.refractivity > 0.) {
			float kr;
			bool outside = dot(currRayDir, normal) < 0;
			fresnel(currRayDir, normal, outside? currObject.refractiveIndex : 1 / currObject.refractiveIndex, kr);
			float3 bias = 0.001 * normal;


			if (kr < 1) {
				float3 refractionDirection = normalize(refract(currRayDir, normal, currObject.refractiveIndex));
				float3 refractionRayOrig = outside ? nextPos - bias : nextPos + bias;
				refracted = currObject.refractivity *(1-kr)* trace(refractionRayOrig, refractionDirection, remainingDepth - 1, currTime, objects, numObjects);
				//if (!outside) {
				//	refracted = make_float3(0, 1, 0);//return make_float3(0, 1, 0);
				//}
			}
			extraReflection = min(1.,kr) * currObject.refractivity;

		}
		if (currObject.reflectivity + extraReflection > 0.) {
			float3 reflectDir = reflect(currRayDir, normal);
			reflected = (currObject.reflectivity + extraReflection )* trace(nextPos + 0.01*reflectDir, reflectDir, remainingDepth - 1, currTime, objects, numObjects);
		}
		float3 color = (1 - currObject.reflectivity - extraReflection - currObject.refractivity) * currObject.color;
		return 10 * (1 / powf(length(nextPos), 1)) * color + reflected + refracted;
	}

}




__global__ void
cudaRender(unsigned int *g_odata, int imgw, int imgh, float currTime)
{

	extern __shared__ uchar4 sdata[];

	int tx = threadIdx.x;
	int ty = threadIdx.y;
	int bw = blockDim.x;
	int bh = blockDim.y;
	int x = blockIdx.x*bw + tx;
	int y = blockIdx.y*bh + ty;

	float sizeNearPlane = 6;
	float sizeFarPlane = 8;
	float distBetweenPlanes = 1.0;


	float3 center = make_float3(imgw / 2.0, imgh / 2.0, 0.);
	float3 distFromCenter = make_float3(x - center.x, y - center.y, 0) *make_float3(1.0f / imgw, 1.0f / imgh, 1.); // coordinate dist from center
	float3 firstPlanePos = sizeNearPlane*distFromCenter;

	float3 secondPlanePos = sizeFarPlane * distFromCenter;
	secondPlanePos.z = -distBetweenPlanes;

	float3 dirVector = normalize(secondPlanePos - firstPlanePos);
	//int out = 0;


	sphereInfo s1 = make_sphereInfo(make_float3(sin(currTime) * 2.0, -3, cos(currTime) * 2 - 15), 1);
	sphereInfo s2 = make_sphereInfo(make_float3(-8, -4, -15), 4);
	sphereInfo s3 = make_sphereInfo(make_float3(2, 3, -40), 6);
	sphereInfo s4 = make_sphereInfo(make_float3(sin(currTime * 0.4)*10 + 2, 0,  cos(currTime*0.4) * 5 - 10), 3);
	planeInfo p1 = make_planeInfo(make_float3(0, -4.0, 0), make_float3(0, -1, 0));
	planeInfo p2 = make_planeInfo(make_float3(0, 10.0, 0), make_float3(0, 1, 0));

	objectInfo objects[6];
	objects[0] = make_objectInfo(sphere, &s1, 0.0, make_float3(1, 0, 0),0,0);
	objects[1] = make_objectInfo(sphere, &s2, 0.1, make_float3(0, 1, 0),0.5,1.5);
	objects[2] = make_objectInfo(plane, &p1, 0.2, make_float3(0, 1, 1),0,0);
	objects[3] = make_objectInfo(sphere, &s3, 0.7, make_float3(1, 1, 1), 0,0);
	objects[4] = make_objectInfo(plane, &p2, 0.0, make_float3(1, 1, 1), 0,0);
	objects[5] = make_objectInfo(sphere, &s4, 0.0, make_float3(1, 0, 0), 1.0,1.5);


	float3 out = 255*trace(firstPlanePos, dirVector, 5, currTime, objects, 6);


	g_odata[y * imgw + x] = rgbToInt(out.x, out.y, out.z);
}

extern "C" void
launch_cudaRender(dim3 grid, dim3 block, int sbytes, unsigned int *g_odata, int imgw, int imgh, float currTime)
{
	cudaRender << < grid, block, sbytes >> >(g_odata, imgw, imgh, currTime);
}
