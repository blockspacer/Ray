#include "hip/hip_runtime.h"
#pragma once
#include "rayHelpers.cu"



//#define USING_POINT_LIGHT
#define STATIC_LIGHT_DIR make_float3(0,1,0)
#define BACKGROUND_COLOR make_float3(53.0/255, 81.0/255, 98.0/255)

hipError_t cuda();
__global__ void kernel() {

}


__device__ bool intersectsSphere(const float3& origin, const float3& dir, const float3 pos, const float rad, float& t) {

	float t0, t1; // solutions for t if the ray intersects 

	float rad2 = powf(rad, 2);

	float3 L = pos - origin;
	float tca = dot(dir, L);
	//if (tca < 0) return false;
	float d2 = dot(L, L) - tca * tca;
	if (d2 > rad2) return false;
	float thc = sqrt(rad2 - d2);
	t0 = tca - thc;
	t1 = tca + thc;

	if (t0 > t1) {
		float temp = t0;
		t0 = t1;
		t1 = temp;
	}

	if (t0 < 0) {
		t0 = t1; // if t0 is negative, let's use t1 instead 
		if (t0 < 0) return false; // both t0 and t1 are negative 
	}
	t = t0;
	return true;
}

// plane normal, plane point, ray start, ray dir, point along line
__device__ bool intersectPlane(const shapeInfo& p, const float3& l0, const float3& l, float& t)
{
	// assuming vectors are all normalized
	float denom = dot(p.normal, l);
	if (denom < -1e-8) {
		float3 p0l0 = p.pos - l0;
		t = dot(p0l0, p.normal) / denom;
		return (t >= 0);
	}
	return false;
}


__device__ bool rayTriangleIntersect(
	float3 orig, float3 dir, float3 v0, const float3& v1, const float3& v2,
	float& t, float& u, float& v)
{
	// compute plane's normal
	float3 v0v1 = v1 - v0;
	float3 v0v2 = v2 - v0;

	//orig = orig - 2*
	//// no need to normalize
	float3 N = cross(v0v1, v0v2); // N 
	float denom = dot(N, N);


	//// Step 1: finding P

	// check if ray and plane are parallel ?
	float NdotRayDirection = dot(N, dir);
	if (fabs(NdotRayDirection) < 0.0001) // almost 0 
		return false; // they are parallel so they don't intersect ! 

	// compute d parameter using equation 2
	float d = dot(N, v0);

	// compute t (equation 3)
	t = (dot(N, orig) + d) / NdotRayDirection;
	// check if the triangle is in behind the ray
	if (t < 0) return false; // the triangle is behind 

	// compute the intersection point using equation 1
	float3 P = orig + t * dir;

	// Step 2: inside-outside test
	float3 C; // vector perpendicular to triangle's plane 

	// edge 0
	float3 edge0 = v1 - v0;
	float3 vp0 = P - v0;
	C = cross(edge0, vp0);
	if (dot(N, C) < 0) return false; // P is on the right side 

	// edge 1
	float3 edge1 = v2 - v1;
	float3 vp1 = P - v1;
	C = cross(edge1, vp1);
	if ((u = dot(N, C)) < 0)  return false; // P is on the right side 

	// edge 2
	float3 edge2 = v0 - v2;
	float3 vp2 = P - v2;
	C = cross(edge2, vp2);
	if ((v = dot(N, C)) < 0) return false; // P is on the right side; 

	u /= denom;
	v /= denom;

	return true; // this ray hits the triangle 
}




__device__ bool RayIntersectsTriangle(float3 rayOrigin,
	float3 rayVector,
	float3 vertex0, float3 vertex1, float3 vertex2,
	float& t, float& u, float& v)
{

	const float EPSILON = 0.001;
	float3 edge1, edge2, h, s, q;
	float a, f;
	edge1 = vertex1 - vertex0;
	edge2 = vertex2 - vertex0;
	h = cross(rayVector, edge2);
	a = dot(edge1, h);
	if (a > -EPSILON && a < EPSILON)
		return false;    // This ray is parallel to this triangle.
	f = 1.0 / a;
	s = rayOrigin - vertex0;
	u = f * dot(s, h);
	//if (u < 0.0 || u > 1.0)
	//	return false;
	q = cross(s, edge1);
	v = f * dot(rayVector, q);
	//if (v < 0.0 || u + v > 1.0)
	//	return false;
	// At this stage we can compute t to find out where the intersection point is on the line.
	t = f * dot(edge2, q);
	if (t > EPSILON && !((u < 0.0 || u > 1.0) || (v < 0.0 || u + v > 1.0))) // ray intersection
	{
		return true;
	}
	else // This means that there is a line intersection but not a ray intersection.
		return false;
}


__device__ void fresnel(const float3& I, const float3& N, const float& ior, float& kr)
{
	float cosi = clamp(-1, 1, dot(I, N));
	float etai = 1, etat = ior;
	if (cosi > 0) { float temp = etai; etai = etat; etat = temp; }
	// Compute sini using Snell's law
	float sint = etai / etat * sqrtf(max(0.f, 1 - cosi * cosi));
	// Total internal reflection
	if (sint >= 1) {
		kr = 1;
	}
	else {
		float cost = sqrtf(max(0.f, 1 - sint * sint));
		cosi = abs(cosi);
		float Rs = ((etat * cosi) - (etai * cost)) / ((etat * cosi) + (etai * cost));
		float Rp = ((etai * cosi) - (etat * cost)) / ((etai * cosi) + (etat * cost));
		kr = (Rs * Rs + Rp * Rp) / 2;
	}

}


__device__ float3 refract(const float3& I, const float3& N, const float& ior)
{
	float cosi = clamp(-1, 1, dot(I, N));
	float etai = 1, etat = ior;
	float3 n = N;
	if (cosi < 0) { cosi = -cosi; }
	else { float temp = etai; etai = etat; etat = temp; n = inverse(N); }
	float eta = etai / etat;
	float k = 1 - eta * eta * (1 - cosi * cosi);
	return eta * I + (eta * cosi - sqrtf(k)) * n;
}

__device__ float3 reflect(const float3& I, const float3& N)
{
	return I - 2 * dot(I, N) * N;
}

struct hitInfo {
	const rayHitInfo* info;
	bool hit;
	//int objectIndex = -1;
	//bool hitMesh = false;
	float3 pos;
	float3 normal;

};

#define LIGHT_POS make_float3(0,10,20)


__device__ hitInfo getHit(float3 currRayPos, float3 currRayDir, const sceneInfo& scene) {
	float closestDist = 1000000;
	float3 normal;
	hitInfo toReturn;
	toReturn.hit = false;


	// mathematical objects
	for (int i = 0; i < scene.numObjects; i++) {
		const objectInfo& curr = scene.objects[i];
		float currDist;

		switch (curr.s) {
		case plane: {
			shapeInfo p1 = curr.shapeData;
			if (intersectPlane(p1, currRayPos, currRayDir, currDist) && currDist < closestDist) {
				closestDist = currDist;
				toReturn.info = &curr.rayInfo;
				normal = p1.normal;
				toReturn.hit = true;
			}

			break;
		}
		case sphere: {
			shapeInfo s1 = curr.shapeData;
			if (intersectsSphere(currRayPos, currRayDir, s1.pos, s1.rad, currDist) && currDist < closestDist) {
				closestDist = currDist;
				float3 nextPos = currRayPos + currDist * currRayDir;
				normal = normalize(nextPos - s1.pos);
				toReturn.info = &curr.rayInfo;
				toReturn.hit = true;

			}
			break;
		}
		}
	}


	// meshes
	for (int i = 0; i < scene.numMeshes; i++) {
		triangleMesh currMesh = scene.meshes[i];

		float tMin = 0;
		float tMax;

		float3 gridPos = (currRayPos - currMesh.bbMin) / currMesh.gridBoxDimensions;
		gridPos = make_float3(floor(gridPos.x), floor(gridPos.y), floor(gridPos.z));

		bool isAlreadyInside = max(gridPos.x, max(gridPos.y, gridPos.z)) < GRID_SIZE && min(gridPos.x, min(gridPos.y, gridPos.z)) >= 0;
		if (isAlreadyInside || (intersectBox(currRayPos, currRayDir, currMesh.bbMin, currMesh.bbMax, tMin, tMax) && tMin < closestDist && tMin > 0)) {

			// engage the GRID
			float3 currPos = currRayPos + (tMin + 0.001)*currRayDir;
			gridPos = (currPos - currMesh.bbMin) / currMesh.gridBoxDimensions;

			int i = 0;
			int stepsBeforeQuit = 100000;
			while (--stepsBeforeQuit >= 0 && max(gridPos.x, max(gridPos.y, gridPos.z)) < GRID_SIZE && min(gridPos.x, min(gridPos.y, gridPos.z)) >= 0) {

				gridPos = make_float3(floor(gridPos.x), floor(gridPos.y), floor(gridPos.z));
				unsigned int gridPosLoc = GRID_POS(gridPos.x, gridPos.y, gridPos.z);

				float t;
				float u;
				float v;
				for (unsigned int j = 0; j < currMesh.gridSizes[gridPosLoc]; j++) {
					unsigned int iPos = currMesh.grid[gridPosLoc][j];
					bool hitTriangle = RayIntersectsTriangle(currRayPos, currRayDir, currMesh.vertices[currMesh.indices[iPos]], currMesh.vertices[currMesh.indices[iPos + 1]], currMesh.vertices[currMesh.indices[iPos + 2]], t, u, v);
					if (hitTriangle && t < closestDist) {
						closestDist = t;
						toReturn.info = &currMesh.rayInfo;

						normal = (1 - v - u) * currMesh.normals[currMesh.indices[iPos]] + u * currMesh.normals[currMesh.indices[iPos + 1]] + v * currMesh.normals[currMesh.indices[iPos + 2]];
						toReturn.hit = true;
						stepsBeforeQuit = 1;
					}
				}

				float3 distFromCorner = currPos - gridPos * currMesh.gridBoxDimensions - currMesh.bbMin;
				float3 distFromOtherCorner = currMesh.gridBoxDimensions - distFromCorner;
				float remainingToHitX = max(-distFromCorner.x / currRayDir.x, distFromOtherCorner.x / currRayDir.x);
				float remainingToHitY = max(-distFromCorner.y / currRayDir.y, distFromOtherCorner.y / currRayDir.y);
				float remainingToHitZ = max(-distFromCorner.z / currRayDir.z, distFromOtherCorner.z / currRayDir.z);
				float minDist = min(remainingToHitX, min(remainingToHitY, remainingToHitZ));

				currPos = currPos + (minDist + 0.01) * currRayDir;
				gridPos = (currPos - currMesh.bbMin) / currMesh.gridBoxDimensions;
			}
		}

	}


	toReturn.normal = normal;
	toReturn.pos = currRayPos + closestDist * currRayDir;
	return toReturn;
}


__device__ float getShadowTerm(const float3 originalPos, const sceneInfo& scene) {
#ifdef USING_POINT_LIGHT
	float3 toLightVec = normalize(LIGHT_POS - originalPos);
#else
	float3 toLightVec = STATIC_LIGHT_DIR;
#endif
	hitInfo hit = getHit(originalPos + 0.01*toLightVec, toLightVec, scene);
#ifdef USING_POINT_LIGHT
	if (!hit.hit || length(hit.pos - originalPos) > length(originalPos - LIGHT_POS)) {
		return 1.;
	}
	return 0.2;
#else 
	if (!hit.hit) {
		return 1.;
	}
	return 0.2;
#endif

	//return objects[hit.objectIndex].refractivity * 0.8 + 0.2;

}


__device__ float3 trace(const float3 currRayPos, const float3 currRayDir, int remainingDepth, const sceneInfo& scene, const hitInfo *prevHitToAddDepthFrom) {
	if (remainingDepth <= 0) {
		return make_float3(0, 0, 0);
	}


	hitInfo hit = getHit(currRayPos, currRayDir, scene);

	if (!hit.hit) {
		return BACKGROUND_COLOR;
	}
	else {

		rayHitInfo info = *hit.info;
		//objectInfo currObject = scene.objects[hit.objectIndex];
		float3 reflected = make_float3(0, 0, 0);
		float3 refracted = make_float3(0, 0, 0);
		float3 nextPos = hit.pos;
		float3 normal = hit.normal;

		float extraReflection = 0;
		float3 extraColor;
		float3 bias = 0.001 * normal;
		//float extraColorSize = 0;
		float prevColorMP = 0;
		float3 extraPrevColor = make_float3(0,0,0);
		if (prevHitToAddDepthFrom && prevHitToAddDepthFrom->info->insideColorDensity > 0.001) {
			prevColorMP = min(1., length(nextPos - currRayPos) * prevHitToAddDepthFrom->info->insideColorDensity);
			extraPrevColor = prevColorMP * prevHitToAddDepthFrom->info->color;
		}

		if (info.refractivity > 0.) {
			float kr;
			bool outside = dot(currRayDir, normal) < 0;
			fresnel(currRayDir, normal, outside ? info.refractiveIndex : 1 / info.refractiveIndex, kr);


			if (kr <= 1) {
				//extraColorSize = outside ? 0 : min(1 - kr, length(nextPos - currRayPos) * info.insideColorDensity);
				float3 refractionDirection = normalize(refract(currRayDir, normal, info.refractiveIndex));
				float3 refractionRayOrig = outside ? nextPos - bias : nextPos + bias;

				refracted = info.refractivity * max(0.,(1 - kr/* - extraColorSize*/)) * trace(refractionRayOrig, refractionDirection, remainingDepth - 1, scene, &hit);
			}
			extraReflection = max(0.,min(1., kr/* - extraColorSize*/)) * info.refractivity;

		}
		if (info.reflectivity + extraReflection > 0.) {
			float3 reflectDir = reflect(currRayDir, normal);
			reflected = (info.reflectivity + extraReflection) * trace(nextPos + bias, reflectDir, remainingDepth - 1, scene, nullptr);
		}
		float3 color = (1.-prevColorMP)*((1 - info.reflectivity - extraReflection - info.refractivity/* + extraColorSize*/) * info.color) + extraPrevColor;
#ifdef USING_POINT_LIGHT
		float3 light_dir = normalize(LIGHT_POS - nextPos) 
		return 1000 * (1 / powf(length(nextPos - LIGHT_POS), 2))  */*getShadowTerm(nextPos + bias, scene)  **/ color + reflected + refracted;
#else
		float3 light_dir = STATIC_LIGHT_DIR;
		float angleFactor = (0.7 + 0.3 * max(0.0, dot(light_dir, normal)));
		return getShadowTerm(nextPos + bias, scene)* 3.0* angleFactor *color + reflected + refracted;
#endif
	}

}

__global__ void
cudaRender(inputPointers pointers, int imgw, int imgh, float currTime, inputStruct input)
{
	extern __shared__ uchar4 sdata[];

	int tx = threadIdx.x;
	int ty = threadIdx.y;
	int bw = blockDim.x;
	int bh = blockDim.y;
	int x = blockIdx.x * bw + tx;
	int y = blockIdx.y * bh + ty;

	float3 forwardV = make_float3(input.forwardX, input.forwardY, input.forwardZ);
	float3 upV = make_float3(input.upX, input.upY, input.upZ);
	float3 rightV = normalize(cross(upV, forwardV));

	float sizeFarPlane = 10;
	float sizeNearPlane = sizeFarPlane * 0.5;
	float3 origin = make_float3(input.currPosX, input.currPosY, input.currPosZ);
	float distFarPlane = 4;
	float distFirstPlane = distFarPlane * 0.5;

	float3 center = make_float3(imgw / 2.0, imgh / 2.0, 0.);
	float3 distFromCenter = ((x - center.x) / imgw) * rightV + ((center.y - y) / imgh) * upV;
	float3 firstPlanePos = (sizeNearPlane * distFromCenter) + origin + (distFirstPlane * forwardV);
	float3 secondPlanePos = (sizeFarPlane * distFromCenter) + (distFarPlane * forwardV) + origin;

	float3 dirVector = normalize(secondPlanePos - firstPlanePos);

	//sceneInfo info = 

	float3 out = 255 * trace(firstPlanePos, dirVector, 5, pointers.scene, nullptr);


	//float3 out = 50*pointers.scene.meshes[0].vertices[10];
	//out = 128*make_float3(pointers.scene.meshes
	pointers.g_odata[y * imgw + x] = rgbToInt(out.x, out.y, out.z);
}
extern "C" void
launch_cudaRender(dim3 grid, dim3 block, int sbytes, inputPointers pointers, int imgw, int imgh, float currTime, inputStruct input)
{

	cudaRender << < grid, block, sbytes >> > (pointers, imgw, imgh, currTime, input);
}

