#include "hip/hip_runtime.h"
#pragma once
#include "rayHelpers.cu"
#include "perlin.h"
#include "hip/hip_runtime.h"



#define USING_SHADOWS
//#define USING_DOUBLE_TAP_SHADOWS
#define USING_PHOTON_MAPPED_SHADOWS
//#define USING_POINT_LIGHT
//#define STATIC_LIGHT_DIR make_float3(0.0,.71,0.71)
//#define LIGHT_POS make_float3(0,2000,2000)

#define STATIC_LIGHT_DIR make_float3(0.0,1,0)
#define LIGHT_POS make_float3(0,2000,0)
//#define AMBIENT_OCCLUSION
//#define VISUALIZE_BOUNDS



hipError_t cuda();
__global__ void kernel() {

}


__device__ float currentTime;
__device__ sceneInfo scene;
__device__ unsigned int* lightImage;
__device__ int imageWidth;
__device__ int imageHeight;
//sceneInfo info;


__device__ __forceinline__ void fresnel(const float3& I, const float3& N, const float& ior, float& kr)
{
	float cosi = clamp(-1, 1, dot(I, N));
	float etai = 1, etat = ior;
	if (cosi > 0) { float temp = etai; etai = etat; etat = temp; }// std::swap(etai, etat);
	// Compute sini using Snell's law
	float sint = etai / etat * sqrtf(max(0.f, 1 - cosi * cosi));
	// Total internal reflection
	if (sint >= 1) {
		kr = 1;
	}
	else {
		float cost = sqrtf(max(0.f, 1 - sint * sint));
		cosi = fabsf(cosi);
		float Rs = ((etat * cosi) - (etai * cost)) / ((etat * cosi) + (etai * cost));
		float Rp = ((etai * cosi) - (etat * cost)) / ((etai * cosi) + (etat * cost));
		kr = (Rs * Rs + Rp * Rp) / 2;
	}
	// As a consequence of the conservation of energy, transmittance is given by:
	// kt = 1 - kr;
}


__device__ __forceinline__ float3 refract(const float3& I, const float3& N, const float& ior)
{
	float cosi = clamp(-1, 1, dot(I, N));
	float etai = 1, etat = ior;
	float3 n = N;
	if (cosi < 0) { cosi = -cosi; }
	else { float temp = etai; etai = etat; etat = temp; n = inverse(N); }
	float eta = etai / etat;
	float k = 1 - eta * eta * (1 - cosi * cosi);
	return eta * I + (eta * cosi - sqrtf(k)) * n;
}

__device__ __forceinline__ float3 reflect(const float3& I, const float3& N)
{
	return I - 2 * dot(I, N) * N;
}

__device__ float3 getDistortion(const float3 normal,const float3 inputPos,const int perlinDepth) {

	float d = 0.01;
	float3 rightDir = make_float3(0, 0, 1);
	float3 otherDir1 = cross(rightDir, normal);
	float3 otherDir2 = cross(otherDir1, normal);

	float axis1;
	float axis2;

	if (fabs(normal.x) > fabs(normal.y) && fabs(normal.x) > fabs(normal.z)) {
		axis1 = inputPos.y;
		axis2 = inputPos.z;

	}
	else if (fabs(normal.y) > fabs(normal.z)) {
		axis1 = inputPos.x;
		axis2 = inputPos.z;
	}
	else {
		axis1 = inputPos.x;
		axis2 = inputPos.y;
	}

	

	float sample1 = perlin2d(axis1,  axis2, 1, perlinDepth);
	float sample2 = perlin2d(axis1 + 10000,  axis2 + 100000, 1, perlinDepth);

	float h1 = perlin2d(axis1 - d, axis2, 1, perlinDepth);
	float h2 = perlin2d(axis1 + d, axis2, 1, perlinDepth);
	float h3 = perlin2d(axis1, axis2 - d, 1, perlinDepth);
	float h4 = perlin2d(axis1, axis2 + d, 1, perlinDepth);

	float d1 =  (h2 - h1) / 2 * d;
	float d2 =  (h4 - h3) / 2 * d;

	//float h1 = 


	return (otherDir1 * d1 + otherDir2 * d2);

}

__device__ bool getTranslatedPos(float3 position, float3 &translatedPos) {
	float3 beforeTranslation = make_float3(LIGHT_BUFFER_WORLD_RATIO * position.x, LIGHT_BUFFER_WORLD_RATIO * position.z, LIGHT_BUFFER_THICKNESS_WORLD_RATIO * position.y);
	translatedPos = beforeTranslation + make_float3(0.5, 0.5, 0.5);
	translatedPos = translatedPos * make_float3(LIGHT_BUFFER_WIDTH, LIGHT_BUFFER_WIDTH, LIGHT_BUFFER_THICKNESS);
	return (translatedPos.x >= 0 && translatedPos.x < LIGHT_BUFFER_WIDTH && translatedPos.y >= 0 && translatedPos.y < LIGHT_BUFFER_WIDTH && translatedPos.z >= 0 && translatedPos.z < LIGHT_BUFFER_THICKNESS);
}

__device__ bool worldPositionToLerpedValue(float3 position, float &value) {
	float3 translatedPos;
	bool OK = getTranslatedPos(position, translatedPos);

	if (OK) {
		int currZ = floor(translatedPos.z) * LIGHT_BUFFER_WIDTH * LIGHT_BUFFER_WIDTH;

		int currY = floor(translatedPos.y);
		int currX = floor(translatedPos.x);
		int nextY = min(currY + 1, LIGHT_BUFFER_WIDTH -1);
		int nextX = min(currX + 1, LIGHT_BUFFER_WIDTH -1);

		int outUL = currZ + (nextY * LIGHT_BUFFER_WIDTH + currX);
		int outLL = currZ + (currY * LIGHT_BUFFER_WIDTH + currX);
		int outUR = currZ + (nextY * LIGHT_BUFFER_WIDTH + nextX);
		int outLR = currZ + (currY * LIGHT_BUFFER_WIDTH + nextX);

		float xFactor = translatedPos.x - floor(translatedPos.x);

		float yFactor = translatedPos.y - floor(translatedPos.y);
		float combinedUpper = lightImage[outUL] * (1.-xFactor) + lightImage[outUR] * (xFactor);
		float combinedDown = lightImage[outLR] * xFactor + lightImage[outLL] * (1. - xFactor);
		float result = combinedUpper* yFactor + (1. - yFactor) * combinedDown;
		value = result;
		return true;
	}
	return false;
}



__device__ bool worldPositionToTextureCoordinate(float3 position, int& out) {
	float3 translatedPos;
	bool res = getTranslatedPos(position, translatedPos);
	out = (((int)(translatedPos.z)) * LIGHT_BUFFER_WIDTH*LIGHT_BUFFER_WIDTH + ((int)translatedPos.y) * LIGHT_BUFFER_WIDTH + (int)(translatedPos.x));
	return res;
}


__device__ hitInfo getHit(const float3 currRayPos,const float3 currRayDir, bool isLightPass) {
	float closestDist = 1000000;
	float3 normal;
	hitInfo toReturn;
	toReturn.hit = false;



	//float3 globalGridPos = (currRayPos - GLOBAL_GRID_MIN) / GLOBAL_GRID_DIMENSIONS;
	//globalGridPos = make_float3(floor(globalGridPos.x), floor(globalGridPos.y), floor(globalGridPos.z));
	//float tMin;
	//float tMax;

	//bool isAlreadyInsideGlobalGrid = max(globalGridPos.x, max(globalGridPos.y, globalGridPos.z)) < GLOBAL_GRID_SIZE && min(globalGridPos.x, min(globalGridPos.y, globalGridPos.z)) >= 0;
	//if (isAlreadyInsideGlobalGrid/* || (intersectBox(currRayPos, currRayDir, GLOBAL_GRID_MIN, GLOBAL_GRID_MAX, tMin, tMax) && tMin > 0)*/) {


		// mathematical objects
		for (int i = 0; i < scene.numObjects; i++) {
			const objectInfo& curr = scene.objects[i];
			float currDist;


			shapeInfo info = curr.shapeData;
			switch (curr.s) {
			case water: {
				shapeInfo otherInfo = info;
				otherInfo.normal = inverse(otherInfo.normal);
				float3 normalToUse = info.normal;
				bool needsToCommunicateInversion = false;
				bool intersected = intersectPlane(info, currRayPos, currRayDir, currDist);
				if (!intersected) {
					intersected = intersectPlane(otherInfo, currRayPos, currRayDir, currDist);
					normalToUse = otherInfo.normal;
					needsToCommunicateInversion = true;
				}

				if (intersected && currDist < closestDist) {
					closestDist = currDist;
					toReturn.info = curr.rayInfo;
					float3 pos = currRayPos + currDist * currRayDir;
					float3 waveInput = pos * 0.3 + make_float3(1 * currentTime + 10000, 10000, 10000);
					float strength = 3000;

					float3 distortion = getDistortion(normalToUse, waveInput, 4);
					normal = normalize(normalToUse + strength * distortion);
					toReturn.hit = true;
					toReturn.normalIsInversed = needsToCommunicateInversion;

				}

				break;
			}
			case plane: {
				if (intersectPlane(info, currRayPos, currRayDir, currDist) && currDist < closestDist) {
					closestDist = currDist;
					toReturn.info = curr.rayInfo;
					normal = info.normal;
					toReturn.hit = true;
				}

				break;
			}
			case sphere: {
				if (length(info.pos - currRayPos) - info.rad < closestDist && intersectsSphere(currRayPos, currRayDir, info.pos, info.rad, currDist) && currDist < closestDist) {
					closestDist = currDist;
					float3 nextPos = currRayPos + currDist * currRayDir;
					normal = normalize(nextPos - info.pos);
					toReturn.info = curr.rayInfo;
					toReturn.hit = true;

				}
				break;
			}
			}
		}


		// meshes
		for (int i = 0; i < scene.numMeshes; i++) {
			triangleMesh currMesh = scene.meshes[i];

			float tMin = 0;
			float tMax;

			float3 gridPos = (currRayPos - currMesh.bbMin) / currMesh.gridBoxDimensions;
			gridPos = make_float3(floor(gridPos.x), floor(gridPos.y), floor(gridPos.z));

			bool isAlreadyInside = max(gridPos.x, max(gridPos.y, gridPos.z)) < GRID_SIZE && min(gridPos.x, min(gridPos.y, gridPos.z)) >= 0;
			if (isAlreadyInside || (intersectBox(currRayPos, currRayDir, currMesh.bbMin, currMesh.bbMax, tMin, tMax) && tMin < closestDist && tMin > 0)) {

				// engage the GRID
				float3 currPos = currRayPos + (tMin + 0.001) * currRayDir;
				gridPos = (currPos - currMesh.bbMin) / currMesh.gridBoxDimensions;

				int stepsBeforeQuit = 1000;
				bool hitGrid = false;
				while (--stepsBeforeQuit >= 0 && max(gridPos.x, max(gridPos.y, gridPos.z)) < GRID_SIZE && min(gridPos.x, min(gridPos.y, gridPos.z)) >= 0) {

					gridPos = make_float3(floor(gridPos.x), floor(gridPos.y), floor(gridPos.z));
					unsigned int gridPosLoc = GRID_POS(gridPos.x, gridPos.y, gridPos.z);

					float t;
					float u;
					float v;
					for (unsigned int j = 0; j < currMesh.gridSizes[gridPosLoc]; j++) {
						unsigned int iPos = currMesh.grid[gridPosLoc][j];
						if (RayIntersectsTriangle(currRayPos, currRayDir, currMesh.vertices[currMesh.indices[iPos]], currMesh.vertices[currMesh.indices[iPos + 1]], currMesh.vertices[currMesh.indices[iPos + 2]], t, u, v) && t < closestDist) {
							closestDist = t;
							toReturn.info = currMesh.rayInfo;

							normal = (1 - v - u) * currMesh.normals[currMesh.indices[iPos]] + u * currMesh.normals[currMesh.indices[iPos + 1]] + v * currMesh.normals[currMesh.indices[iPos + 2]];
							toReturn.hit = true;
							toReturn.pos = currPos + t * currRayDir;
							stepsBeforeQuit = 1;
						}
					}

					float3 distFromCorner = currPos - gridPos * currMesh.gridBoxDimensions - currMesh.bbMin;
					float3 distFromOtherCorner = currMesh.gridBoxDimensions - distFromCorner;
					float remainingToHitX = max(-distFromCorner.x / currRayDir.x, distFromOtherCorner.x / currRayDir.x);
					float remainingToHitY = max(-distFromCorner.y / currRayDir.y, distFromOtherCorner.y / currRayDir.y);
					float remainingToHitZ = max(-distFromCorner.z / currRayDir.z, distFromOtherCorner.z / currRayDir.z);
					float minDist = min(remainingToHitX, min(remainingToHitY, remainingToHitZ)) + 0.01;

					currPos = currPos + minDist * currRayDir;
					gridPos = (currPos - currMesh.bbMin) / currMesh.gridBoxDimensions;
				}
			}

		}
	//}


	toReturn.normal = normal;
	toReturn.pos = currRayPos + closestDist * currRayDir;
	return toReturn;
}



__device__ float getShadowTerm(const float3 originalPos, const float3 normal) {

#ifdef USING_PHOTON_MAPPED_SHADOWS

	float val;
	bool valid = worldPositionToLerpedValue(originalPos, val);
	if (valid) {
		return val*0.01;
	}
	//else {
	//	return 0;
	//}


#endif

#ifndef USING_SHADOWS
	return 1.0;
#endif
	float toReturn;
#ifdef USING_POINT_LIGHT
	float3 toLightVec = normalize(LIGHT_POS - originalPos);
#else 
	float3 toLightVec = STATIC_LIGHT_DIR;
#endif // USING_POINT_LIGHT
	hitInfo hit = getHit(originalPos + 0.001 *toLightVec, toLightVec, false);
#ifdef USING_POINT_LIGHT
	if (!hit.hit || length(hit.pos - originalPos) > length(originalPos - LIGHT_POS)) {
		toReturn = 1.;
	}
	else {
		toReturn = 0.0;
	}
#else 
	if (!hit.hit || length(hit.pos - LIGHT_POS) < 2001.0f) {
		toReturn = 1.;
	}
	else {
		if (hit.info.insideColorDensity > 0.0001) {
			// hack
			toReturn = powf(1. - hit.info.insideColorDensity, length(hit.pos - originalPos));
			toReturn = max(0.,toReturn);
			#ifdef USING_DOUBLE_TAP_SHADOWS
			hit = getHit(hit.pos + 0.01 * toLightVec, toLightVec, false);
			toReturn = (!hit.hit || length(hit.pos - LIGHT_POS) < 2001.0f) ? toReturn : 0.;// max(0., toReturn - hit.info.refractivity);
			#endif

			//toReturn = 1;

		}
		else {
			toReturn = 0.0;

		}
	}
#endif // USING_POINT_LIGHT


	return toReturn;

}


struct Ray {
	float3 currRayPos;
	float3 currRayDir;
	hitInfo prevHitToAddDepthFrom;
	//hitInfo prevPrevHitToAddDepthFrom;
	float totalContributionRemaining = 0.0;
	bool isLightPass = false;
};

__device__ Ray make_ray(float3 pos, float3 dir, hitInfo prevHit, float remaining, bool lightPass) {
	Ray toReturn;
	toReturn.currRayPos = pos;
	toReturn.currRayDir = dir;
	toReturn.prevHitToAddDepthFrom = prevHit;
	toReturn.totalContributionRemaining = remaining;
	toReturn.isLightPass = lightPass;
	return toReturn;

}

#define MAX_RAYS 10

__device__ float3 traceNonRecursive(const float3 initialRayPos, const float3 initialRayDir, int remainingDepth, const hitInfo prevHitToAddDepthFrom, float totalContributionRemaining = 1.0, bool isLightPass = false) {

	Ray firstRay = make_ray(initialRayPos, initialRayDir, prevHitToAddDepthFrom, totalContributionRemaining, isLightPass);
	float3 accumColor = make_float3(0,0,0);

	int currentNbrRays = 1;
	Ray AllRays[MAX_RAYS];
	AllRays[0] = firstRay;

	for (int i = 0; i < remainingDepth && currentNbrRays > 0; i++) {
		for (int j = 0; j < currentNbrRays; j++) {
			Ray currentRay = AllRays[j];

			hitInfo hit = getHit(currentRay.currRayPos, currentRay.currRayDir, isLightPass);
			if (!hit.hit) {
				accumColor = accumColor + currentRay.totalContributionRemaining * AIR_COLOR;// * currentRay.prevHitToAddDepthFrom.info.color;
			}
			else {
				rayHitInfo info = hit.info;

				float3 reflected = make_float3(0, 0, 0);
				float3 refracted = make_float3(0, 0, 0);
				float3 nextPos = hit.pos;
				float3 normal = hit.normal;

				if (hit.info.roughness > 0.001) {
					float3 distortion = getDistortion(normal, nextPos + make_float3(10000, 10000, 10000), 4);
					normal = normalize(normal + distortion * hit.info.roughness);
				}


				float extraReflection = 0;
				float3 extraColor;
				float3 refractBias = 0.002 * normal;
				float3 reflectBias = refractBias;
				bool outside = dot(currentRay.currRayDir, normal) < 0;

				float before = currentRay.totalContributionRemaining;
				float prevColorMP = 1 - powf(1. - currentRay.prevHitToAddDepthFrom.info.insideColorDensity, length(nextPos - currentRay.currRayPos));
				accumColor = accumColor + prevColorMP * currentRay.prevHitToAddDepthFrom.info.color*currentRay.totalContributionRemaining;
				currentRay.totalContributionRemaining *= (1. - prevColorMP);


				if ( info.refractivity* currentRay.totalContributionRemaining > 0.001) {
					float kr = 1.0;
					fresnel(currentRay.currRayDir, normal, outside ? info.refractiveIndex /prevHitToAddDepthFrom.info.refractiveIndex: prevHitToAddDepthFrom.info.refractiveIndex / info.refractiveIndex, kr);

					if (kr < 1) {
						float3 refractionDirection = normalize(refract(currentRay.currRayDir, normal, info.refractiveIndex));
						float3 refractionRayOrig = outside ? nextPos - refractBias : nextPos + refractBias;

						float refracMP = max(0., (1 - kr));
						//refracted = info.refractivity * refracMP * trace(refractionRayOrig, refractionDirection, remainingDepth - 1, outside ^ hit.normalIsInversed ? hit : hitInfo(), totalContributionRemaining * refracMP, isLightPass);
						Ray nextRay = make_ray(refractionRayOrig, refractionDirection, (outside ^ hit.normalIsInversed) ? hit : currentRay.prevHitToAddDepthFrom, info.refractivity * refracMP * currentRay.totalContributionRemaining, isLightPass);
						if (currentNbrRays < MAX_RAYS) {
							AllRays[currentNbrRays] = nextRay;
							currentNbrRays++;
						}

					}

					extraReflection = max(0.0, min(1., kr) * info.refractivity);
				}
				float reflecMP = (info.reflectivity + extraReflection)* currentRay.totalContributionRemaining;
				if ( reflecMP > 0.001 && !isLightPass) {
					float3 reflectDir = reflect(currentRay.currRayDir, normal);
					float3 reflectionOrig = outside ? nextPos + reflectBias : nextPos - reflectBias;

					Ray nextRay = make_ray(reflectionOrig, reflectDir, currentRay.prevHitToAddDepthFrom, reflecMP, isLightPass);
					if (currentNbrRays < MAX_RAYS) {
						AllRays[currentNbrRays] = nextRay;
						currentNbrRays++;
					}
				}

				float colorMultiplier = max(0., (1. - max(0.f, info.reflectivity) - extraReflection - info.refractivity))* currentRay.totalContributionRemaining;
				float3 color = colorMultiplier * info.color;
				float3 light_dir = STATIC_LIGHT_DIR;
				float angleFactor = (0. + 1.0 * max(0.0, dot(light_dir, normal)));


				if (colorMultiplier > 0.001 && !isLightPass) {
					float shadowFactor = getShadowTerm(nextPos + 0.01 * inverse(currentRay.currRayDir), normal);
					accumColor = accumColor + ((0.8 * shadowFactor * angleFactor + 0.2) * 1.0 * color) ;
				}
				else if (isLightPass){

					float strength = max(0., (1. - max(0.f, info.reflectivity) - extraReflection - info.refractivity)) * 100 * before;
					float3 translatedPos;
					bool OK = getTranslatedPos(nextPos, translatedPos);
					if (OK) {
						int currZ = ((int)translatedPos.z) * LIGHT_BUFFER_WIDTH * LIGHT_BUFFER_WIDTH;

						int currY = floor(translatedPos.y);
						int currX = floor(translatedPos.x);
						int nextY = min(currY + 1, imageWidth - 1);
						int nextX = min(currX + 1, imageWidth - 1);
						int nextZ = (min(LIGHT_BUFFER_THICKNESS-1, (int)translatedPos.z + 1)) * LIGHT_BUFFER_WIDTH * LIGHT_BUFFER_WIDTH;

						int outDUL = currZ + (nextY * LIGHT_BUFFER_WIDTH + currX);
						int outDLL = currZ + (currY * LIGHT_BUFFER_WIDTH + currX);
						int outDUR = currZ + (nextY * LIGHT_BUFFER_WIDTH + nextX);
						int outDLR = currZ + (currY * LIGHT_BUFFER_WIDTH + nextX);

						int outUUL = nextZ + (nextY * LIGHT_BUFFER_WIDTH + currX);
						int outULL = nextZ + (currY * LIGHT_BUFFER_WIDTH + currX);
						int outUUR = nextZ + (nextY * LIGHT_BUFFER_WIDTH + nextX);
						int outULR = nextZ + (currY * LIGHT_BUFFER_WIDTH + nextX);

						float xFactor = fmod(translatedPos.x, 1.f);
						float yFactor = fmod(translatedPos.y, 1.f);

						atomicAdd(&lightImage[outDLL], strength * (1. - xFactor) * (1. - yFactor));
						atomicAdd(&lightImage[outDUL], strength * (1. - xFactor) * (yFactor));
						atomicAdd(&lightImage[outDUR], strength * (xFactor) * (yFactor));
						atomicAdd(&lightImage[outDLR], strength * (xFactor) * (1. - yFactor));

						//atomicAdd(&lightImage[outUUL], strength* (1. - xFactor)* (1. - yFactor)* zFactor);
						//atomicAdd(&lightImage[outULL], strength* (1. - xFactor)* (yFactor)* zFactor);
						//atomicAdd(&lightImage[outUUR], strength* (xFactor)* (yFactor)* zFactor);
						//atomicAdd(&lightImage[outULR], strength* (xFactor)* (1. - yFactor)* zFactor);

					}

				}
			}
			AllRays[j] = AllRays[currentNbrRays - 1];
			currentNbrRays--;
		}
	}
	return accumColor;
}



//__device__ float3 trace(const float3 currRayPos, const float3 currRayDir, int remainingDepth, const hitInfo &prevHitToAddDepthFrom, float totalContributionRemaining = 1.0, bool isLightPass = false) {
//
//	hitInfo hit = getHit(currRayPos, currRayDir, isLightPass);
//
//	if (!hit.hit) {
//		return AIR_COLOR;
//	}
//	else {
//
//		rayHitInfo info = hit.info;
//		float3 reflected = make_float3(0, 0, 0);
//		float3 refracted = make_float3(0, 0, 0);
//		float3 nextPos = hit.pos;
//		float3 normal = hit.normal;
//
//		if (hit.info.roughness > 0.0001) {
//			float3 distortion = getDistortion(normal, nextPos + make_float3(10000,10000,10000), 4);
//			normal = normalize(normal + distortion * hit.info.roughness);
//		}
//
//
//		float extraReflection = 0;
//		float3 extraColor;
//		float3 refractBias = 0.001 * normal;
//		float3 reflectBias = 0.0001 * normal;
//		float prevColorMP = 0;
//		float3 extraPrevColor = make_float3(0,0,0);
//		bool outside = dot(currRayDir, normal) < 0;
//
//		if (prevHitToAddDepthFrom.info.insideColorDensity > 0.001) {
//			prevColorMP = 1 - powf(1. - prevHitToAddDepthFrom.info.insideColorDensity, length(nextPos - currRayPos)+1);
//			extraPrevColor = prevColorMP * prevHitToAddDepthFrom.info.color;
//		}
//
//		if (prevColorMP > 0.999 || remainingDepth == 1 || totalContributionRemaining < 0.001)
//			return info.color * (1. - prevColorMP) + extraPrevColor;
//
//		if (info.refractivity* totalContributionRemaining > 0.001) {
//			float kr = 1.0;
//			fresnel(currRayDir, normal, outside ? info.refractiveIndex : 1 / info.refractiveIndex, kr);
//
//
//			if (kr < 1) {
//				float3 refractionDirection = normalize(refract(currRayDir, normal, info.refractiveIndex));
//				float3 refractionRayOrig = outside ? nextPos - refractBias : nextPos + refractBias;
//
//				float refracMP = max(0., (1 - kr));
//				refracted = info.refractivity * refracMP * trace(refractionRayOrig, refractionDirection, remainingDepth - 1,  outside ^ hit.normalIsInversed ? hit : hitInfo(), totalContributionRemaining* refracMP, isLightPass);
//			}
//			extraReflection = max(0.0,min(1., kr) * info.refractivity);
//
//		}
//		if ((info.reflectivity + extraReflection)* totalContributionRemaining > 0.001 && !isLightPass) {
//			float3 reflectDir = reflect(currRayDir, normal);
//			float3 reflectionOrig = outside ? nextPos + reflectBias : nextPos - reflectBias;
//			float reflecMP = info.reflectivity + extraReflection;
//
//			reflected = reflecMP * trace(reflectionOrig, reflectDir, remainingDepth - 1, prevHitToAddDepthFrom, reflecMP*totalContributionRemaining, isLightPass);
//		}
//
//		float colorMultiplier = max(0., (1. - max(0.f,info.reflectivity) - extraReflection - info.refractivity));
//			float3 color = colorMultiplier * info.color;
//			float3 light_dir = STATIC_LIGHT_DIR;
//			float angleFactor = (0. + 1.0 * max(0.0, dot(light_dir, normal)));
//			float shadowFactor = 0;
//		if (!isLightPass) {
//			if (colorMultiplier * (1.-prevColorMP) > 0.1) {
//				shadowFactor = getShadowTerm(nextPos + 0.01 * inverse(currRayDir), normal);
//			}
//			return (1. - prevColorMP) * ((0.8 * shadowFactor * angleFactor + 0.2) * 1.0 * color + reflected + refracted) + extraPrevColor;
//		}
//		else {
//
//			float strength = (1. - prevColorMP) * colorMultiplier * 100;
//			float3 translatedPos;
//			bool OK = getTranslatedPos(nextPos, translatedPos);
//			if (OK) {
//				int currZ = ((int)translatedPos.z) * LIGHT_BUFFER_WIDTH * LIGHT_BUFFER_WIDTH;
//
//				int currY = floor(translatedPos.y);
//				int currX = floor(translatedPos.x);
//				int nextY = min(currY + 1, imageWidth - 1);
//				int nextX = min(currX + 1, imageWidth - 1);
//
//				int outUL = currZ + (nextY * LIGHT_BUFFER_WIDTH + currX);
//				int outLL = currZ + (currY * LIGHT_BUFFER_WIDTH + currX);
//				int outUR = currZ + (nextY * LIGHT_BUFFER_WIDTH + nextX);
//				int outLR = currZ + (currY * LIGHT_BUFFER_WIDTH + nextX);
//
//				float xFactor = fmod(translatedPos.x, 1.f);// -floor(translatedPos.x);
//				float yFactor = fmod(translatedPos.y,1.f);
//
//				atomicAdd(&lightImage[outLL], strength*(1. - xFactor) * (1. - yFactor));
//				atomicAdd(&lightImage[outUL], strength*(1. - xFactor) * (yFactor));
//				atomicAdd(&lightImage[outUR], strength*(xFactor) * (yFactor));
//				atomicAdd(&lightImage[outLR], strength*(xFactor) * (1. - yFactor));
//
//			}
//
//			return currRayPos;
//		}
//	}
//
//}


__global__ void
cudaRender(inputPointers pointers, int imgw, int imgh, float currTime, inputStruct input)
{
	extern __shared__ uchar4 sdata[];

	int tx = threadIdx.x;
	int ty = threadIdx.y;
	int bw = blockDim.x;
	int bh = blockDim.y;
	int x = blockIdx.x * bw + tx;
	int y = blockIdx.y * bh + ty;

	float3 forwardV = make_float3(input.forwardX, input.forwardY, input.forwardZ);
	float3 upV = make_float3(input.upX, input.upY, input.upZ);
	float3 rightV = normalize(cross(upV, forwardV));

	float sizeFarPlane = 10;
	float sizeNearPlane = sizeFarPlane * 0.5;
	float3 origin = make_float3(input.currPosX, input.currPosY, input.currPosZ);
	float distFarPlane = 4;
	float distFirstPlane = distFarPlane * 0.5;

	float3 center = make_float3(imgw / 2.0, imgh / 2.0, 0.);
	float3 distFromCenter = ((x - center.x) / imgw) * rightV + ((center.y - y) / imgh) * upV;
	float3 firstPlanePos = (sizeNearPlane * distFromCenter) + origin + (distFirstPlane * forwardV);
	float3 secondPlanePos = (sizeFarPlane * distFromCenter) + (distFarPlane * forwardV) + origin;

	float3 dirVector = normalize(secondPlanePos - firstPlanePos);


	currentTime = currTime;
	scene = pointers.scene;
	lightImage = pointers.lightImage;
	imageWidth = imgw;
	imageHeight = imgh;
	//float3 out = 255 * 3 * trace(firstPlanePos, dirVector, 10, input.beginMedium, 1.0);
	float3 out = 255 * 3 * traceNonRecursive(firstPlanePos, dirVector, 10, input.beginMedium, 1.0);


	int firstPos = (y * imgw + x) * 4;
	pointers.image1[firstPos] = out.x;
	pointers.image1[firstPos+1] = out.y;
	pointers.image1[firstPos+2] = out.z;
}

__global__ void
cudaLightRender(inputPointers pointers, int imgw, int imgh, float currTime, inputStruct input)
{
	extern __shared__ uchar4 sdata[];

	int tx = threadIdx.x;
	int ty = threadIdx.y;
	int bw = blockDim.x;
	int bh = blockDim.y;
	int x = blockIdx.x * bw + tx;
	int y = blockIdx.y * bh + ty;

	float3 forwardV = STATIC_LIGHT_DIR;
	float3 upV = make_float3(1,0,0);
	float3 rightV = normalize(cross(upV, forwardV));
	upV = cross(forwardV, rightV);

	float2 center = make_float2(imgw / 2.0, imgh / 2.0);
	float3 distFromCenter = ((x - center.x) / imgw) * rightV + ((center.y - y) / imgh) * upV;
	float3 startPos = distFromCenter * LIGHT_PLANE_SIZE + forwardV * 400 ;
	float3 dirVector = inverse(forwardV);


	currentTime = currTime;
	scene = pointers.scene;
	lightImage = pointers.lightImage;
	imageWidth = imgw;
	imageHeight = imgh;

	traceNonRecursive(startPos, dirVector, 6, input.beginMedium, 1.0, true);


}


__global__ void
cudaClear(unsigned int* buffer, int imgw)
{
	extern __shared__ uchar4 sdata[];

	int tx = threadIdx.x;
	int ty = threadIdx.y;
	int tz = threadIdx.z;
	int bw = blockDim.x;
	int bh = blockDim.y;
	int bt = blockDim.z;
	int x = blockIdx.x * bw + tx;
	int y = blockIdx.y * bh + ty;
	int z = blockIdx.z + bt * tz;

	int firstPos = (z * (imgw*imgw) + y * imgw + x);
	buffer[firstPos] = 0;
	//buffer[firstPos + 1] = 0;
	//buffer[firstPos + 2] = 0;
}

extern "C" void
launch_cudaLight(dim3 grid, dim3 block, int sbytes, inputPointers pointers, int imgw, int imgh, float currTime, inputStruct input)
{

	cudaLightRender << < grid, block, sbytes >> > (pointers, imgw, imgh, currTime, input);
}

extern "C" void
launch_cudaClear(dim3 grid, dim3 block, int sbytes, int imgw, unsigned int *buffer)
{

	cudaClear << < grid, block, sbytes >> > (buffer, imgw);
}



extern "C" void
launch_cudaRender(dim3 grid, dim3 block, int sbytes, inputPointers pointers, int imgw, int imgh, float currTime, inputStruct input)
{

	cudaRender << < grid, block, sbytes >> > (pointers, imgw, imgh, currTime, input);
}


