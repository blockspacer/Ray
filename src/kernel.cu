#include "hip/hip_runtime.h"
#pragma once
#include "rayHelpers.cu"
#include "perlin.h"


#define USING_SHADOWS
//#define USING_POINT_LIGHT
//#define SOFT_SHADOWS
#define STATIC_LIGHT_DIR make_float3(0.5,1,0.5)
#define LIGHT_POS make_float3(1000,2000,1000)

//#define AMBIENT_OCCLUSION

//#define VISUALIZE_BOUNDS


//#define USING_SHADOW_FALLOW
#define SHADOW_FALLOW_FACTOR 0.95
#define AIR_DENSITY 0.001
#define AIR_COLOR 1.0*make_float3(53.0/255, 81.0/255, 98.0/255);
hipError_t cuda();
__global__ void kernel() {

}


__device__ float currentTime;
__device__ sceneInfo *scene;
//sceneInfo info;


__device__ bool intersectsSphere(const float3& origin, const float3& dir, const float3 pos, const float rad, float& t) {

	float t0, t1; // solutions for t if the ray intersects 

	float rad2 = powf(rad, 2);

	float3 L = pos - origin;
	float tca = dot(dir, L);
	//if (tca < 0) return false;
	float d2 = dot(L, L) - tca * tca;
	if (d2 > rad2) return false;
	float thc = sqrt(rad2 - d2);
	t0 = tca - thc;
	t1 = tca + thc;

	if (t0 > t1) {
		float temp = t0;
		t0 = t1;
		t1 = temp;
	}

	if (t0 < 0) {
		t0 = t1; // if t0 is negative, let's use t1 instead 
		if (t0 < 0) return false; // both t0 and t1 are negative 
	}
	t = t0;
	return true;
}

// plane normal, plane point, ray start, ray dir, point along line
__device__ bool intersectPlane(const shapeInfo& p, const float3& l0, const float3& l, float& t)
{
	// assuming vectors are all normalized
	float denom = dot(p.normal, l);
	if (denom < -1e-8) {
		float3 p0l0 = p.pos - l0;
		t = dot(p0l0, p.normal) / denom;
		return (t >= 0);
	}
	return false;
}

__device__ bool intersectWater(const shapeInfo& p, const float3& l0, const float3& l, float& t)
{
	// assuming vectors are all normalized
	float denom = dot(p.normal, l);
	if (denom < -1e-8) {
		float3 p0l0 = p.pos - l0;
		t = dot(p0l0, p.normal) / denom;
		return (t >= 0);
	}
	return false;
}

__device__ bool rayTriangleIntersect(
	float3 orig, float3 dir, float3 v0, const float3& v1, const float3& v2,
	float& t, float& u, float& v)
{
	// compute plane's normal
	float3 v0v1 = v1 - v0;
	float3 v0v2 = v2 - v0;

	//orig = orig - 2*
	//// no need to normalize
	float3 N = cross(v0v1, v0v2); // N 
	float denom = dot(N, N);


	//// Step 1: finding P

	// check if ray and plane are parallel ?
	float NdotRayDirection = dot(N, dir);
	if (fabs(NdotRayDirection) < 0.0001) // almost 0 
		return false; // they are parallel so they don't intersect ! 

	// compute d parameter using equation 2
	float d = dot(N, v0);

	// compute t (equation 3)
	t = (dot(N, orig) + d) / NdotRayDirection;
	// check if the triangle is in behind the ray
	if (t < 0) return false; // the triangle is behind 

	// compute the intersection point using equation 1
	float3 P = orig + t * dir;

	// Step 2: inside-outside test
	float3 C; // vector perpendicular to triangle's plane 

	// edge 0
	float3 edge0 = v1 - v0;
	float3 vp0 = P - v0;
	C = cross(edge0, vp0);
	if (dot(N, C) < 0) return false; // P is on the right side 

	// edge 1
	float3 edge1 = v2 - v1;
	float3 vp1 = P - v1;
	C = cross(edge1, vp1);
	if ((u = dot(N, C)) < 0)  return false; // P is on the right side 

	// edge 2
	float3 edge2 = v0 - v2;
	float3 vp2 = P - v2;
	C = cross(edge2, vp2);
	if ((v = dot(N, C)) < 0) return false; // P is on the right side; 

	u /= denom;
	v /= denom;

	return true; // this ray hits the triangle 
}




__device__ bool RayIntersectsTriangle(float3 rayOrigin,
	float3 rayVector,
	float3 vertex0, float3 vertex1, float3 vertex2,
	float& t, float& u, float& v)
{

	const float EPSILON = 0.001;
	float3 edge1, edge2, h, s, q;
	float a, f;
	edge1 = vertex1 - vertex0;
	edge2 = vertex2 - vertex0;
	h = cross(rayVector, edge2);
	a = dot(edge1, h);
	if (a > -EPSILON && a < EPSILON)
		return false;    // This ray is parallel to this triangle.
	f = 1.0 / a;
	s = rayOrigin - vertex0;
	u = f * dot(s, h);
	//if (u < 0.0 || u > 1.0)
	//	return false;
	q = cross(s, edge1);
	v = f * dot(rayVector, q);
	//if (v < 0.0 || u + v > 1.0)
	//	return false;
	// At this stage we can compute t to find out where the intersection point is on the line.
	t = f * dot(edge2, q);
	if (t > EPSILON && !((u < 0.0 || u > 1.0) || (v < 0.0 || u + v > 1.0))) // ray intersection
	{
		return true;
	}
	else // This means that there is a line intersection but not a ray intersection.
		return false;
}


__device__ void fresnel(const float3& I, const float3& N, const float& ior, float& kr)
{
	float cosi = clamp(-1, 1, dot(I, N));
	float etai = 1, etat = ior;
	if (cosi > 0) { float temp = etai; etai = etat; etat = temp; }
	// Compute sini using Snell's law
	float sint = etai / etat * sqrtf(max(0.f, 1 - cosi * cosi));
	// Total internal reflection
	if (sint >= 1) {
		kr = 1;
	}
	else {
		float cost = sqrtf(max(0.f, 1 - sint * sint));
		cosi = abs(cosi);
		float Rs = ((etat * cosi) - (etai * cost)) / ((etat * cosi) + (etai * cost));
		float Rp = ((etai * cosi) - (etat * cost)) / ((etai * cosi) + (etat * cost));
		kr = (Rs * Rs + Rp * Rp) / 2;
	}

}


__device__ float3 refract(const float3& I, const float3& N, const float& ior)
{
	float cosi = clamp(-1, 1, dot(I, N));
	float etai = 1, etat = ior;
	float3 n = N;
	if (cosi < 0) { cosi = -cosi; }
	else { float temp = etai; etai = etat; etat = temp; n = inverse(N); }
	float eta = etai / etat;
	float k = 1 - eta * eta * (1 - cosi * cosi);
	return eta * I + (eta * cosi - sqrtf(k)) * n;
}

__device__ float3 reflect(const float3& I, const float3& N)
{
	return I - 2 * dot(I, N) * N;
}

struct hitInfo {
	rayHitInfo info;
	bool hit = false;
	float3 pos;
	float3 normal;

};




__device__ float3 getWave(float3 pos, float xSpeed, float ySpeed, float xSize, float ySize) {

	return make_float3(1, 1, 1);

	//normal = normalize(info.normal + strength * make_float3(sinf(waveInput.x), 0, sinf(waveInput.z * 0.1)));

}


//float rayMarchUntilIntersection

//#define rayMarchUntilIntersection(FUN, orig, dir, startDist, numIterations) \
//float t = startDist; \
//for (int i = 0; i < numIterations; i++) { \
//	AA
//
//}

__device__ hitInfo getHit(const float3 currRayPos,const float3 currRayDir) {
	float closestDist = 1000000;
	float3 normal;
	hitInfo toReturn;
	toReturn.hit = false;


	// mathematical objects
	for (int i = 0; i < scene->numObjects; i++) {
		const objectInfo& curr = scene->objects[i];
		float currDist;


		shapeInfo info = curr.shapeData;
		switch (curr.s) {
		case water: {
			if (intersectPlane(info, currRayPos, currRayDir, currDist) && currDist < closestDist) {

				closestDist = currDist;
				toReturn.info = curr.rayInfo;
				float3 waveInput = (currRayPos + currDist * currRayDir)*0.5 + make_float3(3*currentTime + 10000, 10000,10000);
				float strength = 2000.;

				float d = 0.01;
				float h1 = perlin2d(waveInput.x - d, waveInput.z, 1, 3);
				float h2 = perlin2d(waveInput.x + d, waveInput.z, 1, 3);
				float h3 = perlin2d(waveInput.x, waveInput.z - d, 1, 3);
				float h4 = perlin2d(waveInput.x, waveInput.z + d, 1, 3);

				// derivatives
				float d1 = (h2 - h1) / 2*d; 
				float d2 = (h4 - h3) / 2*d;

				//float3 distortion = make_float3(perlin2d(waveInput.x+10000, 0.0, 1, 1)*2* - 1., 0, perlin2d(waveInput.z + 20000, waveInput.x + 30000, 1, 1)*2.-1.);
				float3 rightDir = make_float3(0, 0, 1);
				float3 otherDir1 = cross(rightDir, info.normal);
				float3 otherDir2 = cross(otherDir1, info.normal);

				//otherDir1 = dot(currRayDir, otherDir1) > 0.1 ? inverse(otherDir1) : otherDir1;
				//otherDir2 = dot(currRayDir, otherDir2) > 0.1 ? inverse(otherDir2) : otherDir2;

				float3 distortion = (otherDir1 * d1 +otherDir2 * d2);

				//distortion = dot(distortion, currRayDir) <= -0.0000001 ? distortion : inverse(distortion);

				//distortion = distortion * make_float3(currRayDir.x, currRayDir.y, currRayDir.z);

				normal = normalize(info.normal + strength*distortion);
				//normal = normalize(info.normal + strength*make_float3(sinf(waveInput.x+currentTime), 0, sinf(waveInput.z + currentTime)));
				//normal = normalize(info.normal + strength * make_float3(sinf(waveInput.x + currentTime), 0, 0));
				//normal = make_float3(currRayDir.x > 0 ? -abs(normal.x) : abs(normal.x), currRayDir.y > 0 ? -abs(normal.y) : abs(normal.y), currRayDir.z > 0 ? -abs(normal.z) : abs(normal.z));
				//normal = dot(currRayDir, normal) < 0. ? inverse(normal) + 2 * normal : normal;

				toReturn.hit = true;
			}

			break;
		}
		case plane: {
			if (intersectPlane(info, currRayPos, currRayDir, currDist) && currDist < closestDist) {
				closestDist = currDist;
				toReturn.info = curr.rayInfo;
				normal = info.normal;
				toReturn.hit = true;
			}

			break;
		}
		case sphere: {
			if (intersectsSphere(currRayPos, currRayDir, info.pos, info.rad, currDist) && currDist < closestDist) {
				closestDist = currDist;
				float3 nextPos = currRayPos + currDist * currRayDir;
				normal = normalize(nextPos - info.pos);
				toReturn.info = curr.rayInfo;
				toReturn.hit = true;

			}
			break;
		}
		}
	}


	// meshes
	for (int i = 0; i < scene->numMeshes; i++) {
		triangleMesh currMesh = scene->meshes[i];

		float tMin = 0;
		float tMax;

		float3 gridPos = (currRayPos - currMesh.bbMin) / currMesh.gridBoxDimensions;
		gridPos = make_float3(floor(gridPos.x), floor(gridPos.y), floor(gridPos.z));

		bool isAlreadyInside = max(gridPos.x, max(gridPos.y, gridPos.z)) < GRID_SIZE && min(gridPos.x, min(gridPos.y, gridPos.z)) >= 0;
		/*intersectsSphere(currRayPos, currRayDir, 0.5 * (currMesh.bbMin + currMesh.bbMax), currMesh.rad, tMin);*/
		if (isAlreadyInside || intersectsSphere(currRayPos, currRayDir, 0.5 * (currMesh.bbMin + currMesh.bbMax), currMesh.rad, tMin) && (intersectBox(currRayPos, currRayDir, currMesh.bbMin, currMesh.bbMax, tMin, tMax) && tMin < closestDist && tMin > 0)) {

			// engage the GRID
			float3 currPos = currRayPos + (tMin + 0.001)*currRayDir;
			gridPos = (currPos - currMesh.bbMin) / currMesh.gridBoxDimensions;

			int stepsBeforeQuit = 1000;
			while (--stepsBeforeQuit >= 0 && max(gridPos.x, max(gridPos.y, gridPos.z)) < GRID_SIZE && min(gridPos.x, min(gridPos.y, gridPos.z)) >= 0) {

				gridPos = make_float3(floor(gridPos.x), floor(gridPos.y), floor(gridPos.z));
				unsigned int gridPosLoc = GRID_POS(gridPos.x, gridPos.y, gridPos.z);

				float t;
				float u;
				float v;
				for (unsigned int j = 0; j < currMesh.gridSizes[gridPosLoc]; j++) {
					unsigned int iPos = currMesh.grid[gridPosLoc][j];
					bool hitTriangle = RayIntersectsTriangle(currRayPos, currRayDir, currMesh.vertices[currMesh.indices[iPos]], currMesh.vertices[currMesh.indices[iPos + 1]], currMesh.vertices[currMesh.indices[iPos + 2]], t, u, v);
					if (hitTriangle && t < closestDist) {
						closestDist = t;
						toReturn.info = currMesh.rayInfo;

						normal = (1 - v - u)* currMesh.normals[currMesh.indices[iPos]] + u * currMesh.normals[currMesh.indices[iPos + 1]] + v * currMesh.normals[currMesh.indices[iPos + 2]];
						toReturn.hit = true;
						stepsBeforeQuit = 1;
					}
				}

				float3 distFromCorner = currPos - gridPos * currMesh.gridBoxDimensions - currMesh.bbMin;
				float3 distFromOtherCorner = currMesh.gridBoxDimensions - distFromCorner;
				float remainingToHitX = max(-distFromCorner.x / currRayDir.x, distFromOtherCorner.x / currRayDir.x);
				float remainingToHitY = max(-distFromCorner.y / currRayDir.y, distFromOtherCorner.y / currRayDir.y);
				float remainingToHitZ = max(-distFromCorner.z / currRayDir.z, distFromOtherCorner.z / currRayDir.z);
				float minDist = min(remainingToHitX, min(remainingToHitY, remainingToHitZ)) + 0.01;

				currPos = currPos + minDist * currRayDir;
				gridPos = (currPos - currMesh.bbMin) / currMesh.gridBoxDimensions;
			}
		}

	}


	toReturn.normal = normal;
	toReturn.pos = currRayPos + closestDist * currRayDir;
	return toReturn;
}



__device__ float getShadowTerm(const float3 originalPos, const float3 normal) {

#ifndef USING_SHADOWS
	return 1.0;
#endif
	float toReturn;
#ifndef SOFT_SHADOWS
#ifdef USING_POINT_LIGHT
	float3 toLightVec = normalize(LIGHT_POS - originalPos);
#else 
	float3 toLightVec = STATIC_LIGHT_DIR;
#endif // USING_POINT_LIGHT
	hitInfo hit = getHit(originalPos + 0.001 *toLightVec, toLightVec);
#ifdef USING_POINT_LIGHT
	if (!hit.hit || length(hit.pos - originalPos) > length(originalPos - LIGHT_POS)) {
		toReturn = 1.;
	}
	else {
		toReturn = 0.0;
	}
#else 
	if (!hit.hit || length(hit.pos - LIGHT_POS) < 2001.0f) {
		toReturn = 1.;
	}
	else {
		#ifdef USING_SHADOW_FALLOW
			toReturn = min(1.f, 1.-powf(SHADOW_FALLOW_FACTOR,length(hit.pos - originalPos)));
		#else
		toReturn = 0.0;
		#endif
	}
#endif // USING_POINT_LIGHT

#else // if soft shadows

	int totalHits = 0;
	for (int x = 0; x < 2; x++) {
		for (int y = 0; y < 2; y++) {
			float3 toLightVec = normalize(STATIC_LIGHT_DIR + 0.05*make_float3(x,0,y));
			hitInfo hit = getHit(originalPos + 0.001 * toLightVec, toLightVec);
			if (!hit.hit || length(hit.pos - originalPos) > length(originalPos - LIGHT_POS))
				totalHits++;
		}
	}

	toReturn = ((float)totalHits / 4);
#endif  // soft shadows

#ifdef AMBIENT_OCCLUSION
	float3 rightDir = make_float3(0, 0, 1);
	float3 otherDir1 = cross(rightDir, normal);
	float3 otherDir2 = cross(otherDir1, normal);

	float maxLen = 0.2;
	float minDist = maxLen;
	for (int x = -5; x < 5; x++) {
		for (int y = -5; y < 5; y++) {
			float3 currDir = normal + otherDir1 * x + otherDir2 * y;
			hitInfo hit = getHit(originalPos + 0.001 * currDir, currDir);
			minDist = min(minDist, length(hit.pos - originalPos));
			//if (!hit.hit || length(hit.pos - originalPos) > maxLen)
			//	totalHits+;
		}
	}

	toReturn *= (minDist / maxLen);

#endif

	return toReturn;

}


__device__ float3 trace(const float3 currRayPos, const float3 currRayDir, int remainingDepth, const hitInfo prevHitToAddDepthFrom) {
	if (remainingDepth <= 0) {
		return AIR_COLOR;
	}


	hitInfo hit = getHit(currRayPos, currRayDir);

	if (!hit.hit) {
		return AIR_COLOR;
	}
	else {

		rayHitInfo info = hit.info;
		//objectInfo currObject = scene.objects[hit.objectIndex];
		float3 reflected = make_float3(0, 0, 0);
		float3 refracted = make_float3(0, 0, 0);
		float3 nextPos = hit.pos;
		float3 normal = hit.normal;

		float extraReflection = 0;
		float3 extraColor;
		float3 refractBias = 0.0005 * normal;
		float3 reflectBias = 0.001 * normal;
		float prevColorMP = 0;
		float3 extraPrevColor = make_float3(0,0,0);
		bool outside = dot(currRayDir, hit.normal) < 0;

		if (prevHitToAddDepthFrom.hit && prevHitToAddDepthFrom.info.insideColorDensity > 0.001) {
			prevColorMP = 1 - powf(1. - prevHitToAddDepthFrom.info.insideColorDensity, length(nextPos - currRayPos));
			extraPrevColor = prevColorMP * prevHitToAddDepthFrom.info.color;
		}
		else if (!prevHitToAddDepthFrom.hit){
			prevColorMP = 1 - powf(1. - AIR_DENSITY, length(nextPos - currRayPos));
			extraPrevColor = prevColorMP * AIR_COLOR;
		}

		if (info.refractivity > 0.0001) {
			float kr;
			fresnel(currRayDir, normal, outside ? info.refractiveIndex : 1 / info.refractiveIndex, kr);


			if (kr <= 1) {
				float3 refractionDirection = normalize(refract(currRayDir, normal, info.refractiveIndex));
				float3 refractionRayOrig = outside ? nextPos - refractBias : nextPos + refractBias;

				refracted = info.refractivity * max(0.,(1 - kr)) * trace(refractionRayOrig, refractionDirection, remainingDepth - 1,  hit);
			}
			extraReflection = max(0.,min(1., kr) * info.refractivity);

		}
		if (info.reflectivity + extraReflection > 0.00001) {
			float3 reflectDir = reflect(currRayDir, normal);
			float3 reflectionOrig = outside ? nextPos + reflectBias : nextPos - reflectBias;// nextPos + reflectNormal * 0.001;// : nextPos - reflectNormal * 0.01;//0.01;


			reflected = (info.reflectivity + extraReflection) * trace(reflectionOrig, reflectDir, remainingDepth - 1, hitInfo());
		}
		float3 color = ((1 - info.reflectivity - extraReflection - info.refractivity) * info.color);
#ifdef USING_POINT_LIGHT
		float3 light_dir = normalize(LIGHT_POS - nextPos);
		return 10000 * (1 / powf(length(nextPos - LIGHT_POS), 2)) *(0.2 + 0.8*getShadowTerm(nextPos + bias, scene)) * color + reflected + refracted;
#else
		float3 light_dir = STATIC_LIGHT_DIR;
		float angleFactor = (0.0 + 1.0 * max(0.0, dot(light_dir, normal)));
		return (1. - prevColorMP) * ((0.8*getShadowTerm(nextPos + 0.01*inverse(currRayDir),normal)* angleFactor + 0.2)* 1.0 *color + reflected + refracted) + extraPrevColor;
#endif
	}

}

__global__ void
cudaRender(inputPointers pointers, int imgw, int imgh, float currTime, inputStruct input)
{
	extern __shared__ uchar4 sdata[];

	int tx = threadIdx.x;
	int ty = threadIdx.y;
	int bw = blockDim.x;
	int bh = blockDim.y;
	int x = blockIdx.x * bw + tx;
	int y = blockIdx.y * bh + ty;

	float3 forwardV = make_float3(input.forwardX, input.forwardY, input.forwardZ);
	float3 upV = make_float3(input.upX, input.upY, input.upZ);
	float3 rightV = normalize(cross(upV, forwardV));

	float sizeFarPlane = 10;
	float sizeNearPlane = sizeFarPlane * 0.5;
	float3 origin = make_float3(input.currPosX, input.currPosY, input.currPosZ);
	float distFarPlane = 4;
	float distFirstPlane = distFarPlane * 0.5;

	float3 center = make_float3(imgw / 2.0, imgh / 2.0, 0.);
	float3 distFromCenter = ((x - center.x) / imgw) * rightV + ((center.y - y) / imgh) * upV;
	float3 firstPlanePos = (sizeNearPlane * distFromCenter) + origin + (distFirstPlane * forwardV);
	float3 secondPlanePos = (sizeFarPlane * distFromCenter) + (distFarPlane * forwardV) + origin;

	float3 dirVector = normalize(secondPlanePos - firstPlanePos);


	currentTime = currTime;
	scene = &pointers.scene;
	float3 out = 255 * trace(firstPlanePos, dirVector, 10, hitInfo())*3;

	pointers.g_odata[y * imgw + x] = rgbToInt(out.x, out.y, out.z);
}
extern "C" void
launch_cudaRender(dim3 grid, dim3 block, int sbytes, inputPointers pointers, int imgw, int imgh, float currTime, inputStruct input)
{

	cudaRender << < grid, block, sbytes >> > (pointers, imgw, imgh, currTime, input);
}

