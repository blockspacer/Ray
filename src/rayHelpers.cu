#include "hip/hip_runtime.h"
#pragma once
#include "hip/hip_runtime.h"
#include "vector_functions.h"
#include "common_functions.h"
#include "math_functions.h"
#include "sharedStructs.h"
#include <stdlib.h>
#include <stdio.h>

// clamp x to range [a, b]
inline __device__ float clamp(float x, float a, float b)
{
	return max(a, min(b, x));
}

// convert floating point rgb color to 8-bit integer
inline __device__ int rgbToInt(float r, float g, float b)
{
	r = clamp(r, 0.0f, 255.0f);
	g = clamp(g, 0.0f, 255.0f);
	b = clamp(b, 0.0f, 255.0f);
	return (int(b) << 16) | (int(g) << 8) | int(r);
}



inline __device__ float3 rotateAngleAxis(const float3 vector, const float angleDeg, const float3& axis) 
{
	double S, C;
	sincos(angleDeg, &S, &C);
	//FMath::SinCos(&S, &C, FMath::DegreesToRadians(angleDeg));

	const float XX = axis.x * axis.x;
	const float YY = axis.y * axis.y;
	const float ZZ = axis.z * axis.z;

	const float XY = axis.x * axis.y;
	const float YZ = axis.y * axis.z;
	const float ZX = axis.z * axis.x;

	const float XS = axis.x * S;
	const float YS = axis.y * S;
	const float ZS = axis.z * S;

	const float OMC = 1.f - C;

	return make_float3(
		(OMC * XX + C) * vector.x + (OMC * XY - ZS) * vector.y + (OMC * ZX + YS) * vector.z,
		(OMC * XY + ZS) * vector.x + (OMC * YY + C) * vector.y + (OMC * YZ - XS) * vector.z,
		(OMC * ZX - YS) * vector.x + (OMC * YZ + XS) * vector.y + (OMC * ZZ + C) *vector.z 
	);
}



