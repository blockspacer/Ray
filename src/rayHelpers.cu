#include "hip/hip_runtime.h"
#pragma once
#include "hip/hip_runtime.h"
#include "vector_functions.h"
#include "common_functions.h"
#include "math_functions.h"
#include "sharedStructs.h"
#include <stdlib.h>
#include <stdio.h>

// clamp x to range [a, b]
inline __device__ float clamp(float x, float a, float b)
{
	return max(a, min(b, x));
}

// convert floating point rgb color to 8-bit integer
inline __device__ int rgbToInt(float r, float g, float b)
{
	r = clamp(r, 0.0f, 255.0f);
	g = clamp(g, 0.0f, 255.0f);
	b = clamp(b, 0.0f, 255.0f);
	return (int(b) << 16) | (int(g) << 8) | int(r);
}



inline __device__ float3 rotateAngleAxis(const float3 vector, const float angleDeg, const float3& axis) 
{
	double S, C;
	sincos(angleDeg, &S, &C);

	const float XX = axis.x * axis.x;
	const float YY = axis.y * axis.y;
	const float ZZ = axis.z * axis.z;

	const float XY = axis.x * axis.y;
	const float YZ = axis.y * axis.z;
	const float ZX = axis.z * axis.x;

	const float XS = axis.x * S;
	const float YS = axis.y * S;
	const float ZS = axis.z * S;

	const float OMC = 1.f - C;

	return make_float3(
		(OMC * XX + C) * vector.x + (OMC * XY - ZS) * vector.y + (OMC * ZX + YS) * vector.z,
		(OMC * XY + ZS) * vector.x + (OMC * YY + C) * vector.y + (OMC * YZ - XS) * vector.z,
		(OMC * ZX - YS) * vector.x + (OMC * YZ + XS) * vector.y + (OMC * ZZ + C) *vector.z 
	);
}


inline __device__ float rand(float2 co) {

	float val= sinf(dot(make_float2(co.x, co.y), make_float2(12.9898, 78.233)) * 43758.5453);
	return val - floor(val);
}

